#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void Bellmanor::copydata(int s,vector<edge>&edges,int nodenum){
};
void Bellmanor::dellocate(){
};
void Bellmanor::allocate(int maxn,int maxedge){
}
void Bellmanor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void Bellmanor::updatE(vector<int>esigns)
{
	
}
void Bellmanor::updatS(vector<vector<pair<int,int>>>&stpair)
{
	cout<<"inasd asd"<<endl;
	L[0]=LY1;
	L[1]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	int count=0;
	ncount=L[0]*S[0]+L[1]*S[1];
	for(int i=0;i<nodenum*ncount;i++)
		d[i]=100000,p[i]=-1;
	int woffid=0;
	for(int h=0;h<stpair.size();h++)
		{
		for(int k=0;k<L[h];k++)
			{
			for(int j=0;j<stpair[h].size();j++)
				{
				 d[count*nodenum+stpair[h][j].first]=0;
				 count++;
				}
			}
		}
	cout<<"here it is "<<endl;
	for(int i=1;i<NF.size();i++)
		NF[i]=L[i-1]*S[i-1];
	nodeoff[0]=0;
	nodeoff[1]=S[0]*L[0]*nodenum;
	leveloff[0]=0;
	leveloff[1]=L[0]*edges.size();
	size[0]=edges.size()*L[0]*S[0];
	size[1]=edges.size()*L[1]*S[1];
	cout<<"asd"<<endl;
	cout<<"ncount is "<<count <<endl;
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
	cout<<"out!!!!"<<endl;
}
void Bellmanor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	cout<<"init bellmanor"<<endl;
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	has=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	w=new int[edges.size()*LY];
	m1=new int;
	m2=new int;
	*m1=0,*m2=0;
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	vector<vector<int>>rus(nodenum,vector<int>());
	vector<vector<int>>ruw(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			rus[t].push_back(s);
			ruw[t].push_back(i);
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	mm=0;
	for(int i=0;i<rus.size();i++)
		if(rus[i].size()>mm)mm=rus[i].size();
	rudu=new int[nodenum*mm*LY];
	rudw=new int[nodenum*mm*LY];
	for(int k=0;k<LY;k++)
		{
		int off=k*nodenum*mm;
		for(int i=0;i<nodenum;i++)
		{
			for(int j=0;j<mm;j++)
				if(j<rus[i].size())
					rudu[off+i*mm+j]=rus[i][j];
				else
					rudu[off+i*mm+j]=INT_MAX;
			for(int j=0;j<mm;j++)
				if(j<rus[i].size())
					rudw[off+i*mm+j]=esigns[k][ruw[i][j]];
				else
					rudw[off+i*mm+j]=-1;
		}
		}
	/*for(int i=0;i<nodenum;i++)
		{for(int j=0;j<mm;j++)
			{
			if(rudu[i*mm+j]<INT_MAX)
				cout<<rudu[i*mm+j]<<" ";
			}
		cout<<endl;
		}*/
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	int cc=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			w[cc++]=esigns[k][i];
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_w,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_m1,sizeof(int));
	hipMalloc((void**)&dev_m2,sizeof(int));
	hipMalloc((void**)&dev_rudu,mm*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_rudw,mm*LY*nodenum*sizeof(int));
	
	if(dev_d==NULL) {
		printf("couldn't allocate %d int's.\n");
	}
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_w,w,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rudu,rudu,mm*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rudw,rudw,mm*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice);
	//cout<<nodenum<<endl;
};
Bellmanor::Bellmanor():L(2,0),S(2,0),NF(2,0),nodeoff(2,0),leveloff(2,0),size(2,0)
{
};
__global__ void bellmanhigh(int *st,int *te,int *d,int *has,int *w,int E,int N,int size,int *m,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight<d[t+off])  
		{
			d[t+off]=weight+d[s+off];
			//has[t+off]=round;
			*m=1;
		}
}
__global__ void bellmandu(int *rudu,int*rudw,int *d,int*p,int N,int size,int sizeoff,int leveloff,int ye,int ly,int mm)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=size)return;
	int lyy=i/(ye*N)+leveloff;
	int yee=(i%(ye*N))/N;
	int off=lyy*N*ye+yee*N;
	int roff=(i%N+lyy*N)*mm;
	i+=sizeoff;
	int dm=d[i];
	int mark=-1;
	for(int k=0;k<mm;k++)
		if(rudu[roff+k]<INT_MAX)
		{
			int node=rudu[roff+k]+off;
			if(rudw[roff+k]<0)continue;
			if(dm>d[node]+rudw[roff+k])
				{dm=d[node]+rudw[roff+k];
				mark=k;
				}
		}
	if(d[i]>dm)
		d[i]=dm,p[i]=k;
	//if(sizeoff>0)
		//d[i]=0;
}
__global__ void color(int *st,int *te,int *d,int *pre,int *has,int *w,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight==d[t+off])
		pre[t+off]=s+off;
}
vector<vector<int>> Bellmanor::routalg(int s,int t,int bw)
{
	cout<<"inbellman"<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	*m1=1;
	*m2=1;
	int round=1;
	cout<<"fuck wx!"<<endl;
	int flag1=0,flag2=0;
	int cc=0;
	/*while(*m2==1||*m1==1)
	{
		*m2=0,*m1=0;
		hipMemcpyAsync(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice,stream1);
		bellmanhigh<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[1],dev_m2,round,leveloff[1],nodeoff[1],S[1],L[1]);
		hipMemcpyAsync(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice,stream0);
		bellmanhigh<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[0],dev_m2,round,leveloff[0],nodeoff[0],S[0],L[0]);
		color<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[1],round,leveloff[1],nodeoff[1],S[1],L[1]);
		hipMemcpyAsync(m2,dev_m2,sizeof(int),hipMemcpyDeviceToHost,stream1);
		color<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[0],round,leveloff[0],nodeoff[0],S[0],L[0]);
		hipMemcpyAsync(m1,dev_m1,sizeof(int),hipMemcpyDeviceToHost,stream0);
		hipStreamSynchronize(stream1);
		hipStreamSynchronize(stream0);
	}*/
	cout<<"here it is "<<endl;
	int size0=nodenum*L[0]*S[0];
	int size1=nodenum*L[1]*S[1];
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	cout<<"l1:"<<L[1]<<endl;
	for(int i=0;i<WD+1;i++)
	{
		bellmandu<<<size0/1024+1,1024,0,stream0>>>(dev_rudu,dev_rudw,dev_d,nodenum,size0,0,0,S[0],L[0],mm);
		bellmandu<<<size1/1024+1,1024,0,stream1>>>(dev_rudu,dev_rudw,dev_d,nodenum,size1,size0,0,S[1],L[1],mm);
	}
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream0);
	hipMemcpy(d,dev_d,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	/*for(int j=0;j<200;j++)
		{for(int i=0;i<nodenum;i++)
			cout<<d[i+j*nodenum]<<" ";
		cout<<endl;
		}*/
	cout<<endl;
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	vector<vector<int>>result(LY,vector<int>());
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	hipFree(dev_w);
	cout<<"before return"<<endl;
	return result;
};