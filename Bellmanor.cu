#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void Bellmanor::copydata(int s,vector<edge>&edges,int nodenum){
};
void Bellmanor::dellocate(){
};
void Bellmanor::allocate(int maxn,int maxedge){
}
void Bellmanor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void Bellmanor::updatE(vector<int>esigns)
{
	
}
void Bellmanor::updatS(vector<vector<pair<int,int>>>&stpair)
{
	L[0]=LY1;
	L[1]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	int count=0;
	ncount=L[0]*S[0]+L[1]*S[1];
	for(int i=0;i<nodenum*ncount;i++)
		d[i]=INT_MAX/2,p[i]=-1;
	int woffid=0;
	for(int h=0;h<stpair.size();h++)
		{
		for(int k=0;k<L[h];k++)
			{
				for(int i=0;i<S[h];i++)
				{
					for(int j=0;j<stpair[h].size();j++)
						{
						 d[count*nodenum+stpair[h][i].first]=0;
						 count++;
						}
				}
			}
		}
	for(int j=0;j<4;j++)
	{
		for(int i=0;i<nodenum;i++)
			cout<<d[i+j*nodenum]<<" ";
		cout<<endl;
	}
	for(int i=1;i<NF.size();i++)
		NF[i]=L[i-1]*S[i-1];
	nodeoff[0]=0;
	nodeoff[1]=S[0]*L[0]*nodenum;
	leveloff[0]=0;
	leveloff[1]=L[0]*edges.size();
	size[0]=edges.size()*L[0]*S[0];
	size[1]=edges.size()*L[1]*S[1];
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void Bellmanor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	cout<<"init bellmanor"<<endl;
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	has=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	w=new int[edges.size()*LY];
	m1=new int;
	m2=new int;
	*m1=0,*m2=0;
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	int cc=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			w[cc++]=esigns[k][i];
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_has,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_w,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_m1,sizeof(int));
	hipMalloc((void**)&dev_m2,sizeof(int));
	if(dev_d==NULL) {
		printf("couldn't allocate %d int's.\n");
	}
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_w,w,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_has,has,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice);
	cout<<nodenum<<endl;
};
Bellmanor::Bellmanor():L(2,0),S(2,0),NF(2,0),nodeoff(2,0),leveloff(2,0),size(2,0)
{
};
__global__ void bellmanhigh(int *st,int *te,int *d,int *has,int *w,int E,int N,int size,int *m,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight<d[t+off])  
		{
			d[t+off]=weight+d[s+off];
			has[t+off]=round;
		}
}
__global__ void color(int *st,int *te,int *d,int *pre,int *has,int *w,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight==d[t+off])
		pre[t+off]=s+off;
}
vector<vector<int>> Bellmanor::routalg(int s,int t,int bw)
{
	cout<<"inbellman"<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	*m1=1;
	*m2=1;
	int round=1;
	cout<<"asdaasadadadasd"<<endl;
	hipStream_t stream0,stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	int flag1=0,flag2=0;
	int cc=0;
	//while(*m2==1)
	for(int i=0;i<10;i++)
	{
		
		bellmanhigh<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[1],dev_m2,round,leveloff[1],nodeoff[1],S[1],L[1]);
		bellmanhigh<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[0],dev_m2,round,leveloff[0],nodeoff[0],S[0],L[0]);
		color<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[1],round,leveloff[1],nodeoff[1],S[1],L[1]);
		color<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[0],round,leveloff[0],nodeoff[0],S[0],L[0]);
		/**m2=0;
		*m1=0;
		if(flag2==0&&cc)hipMemcpyAsync(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice,stream1);
		if(flag2==0)bellmanhigh<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[1],dev_m2,round,leveloff[1],nodeoff[1],S[1],L[1]);
		if(flag1==0)hipMemcpyAsync(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice,stream0);
		if(flag1==0)bellmanhigh<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[0],dev_m2,round,leveloff[0],nodeoff[0],S[0],L[0]);
	 	//color<<<size[1]/1024+1,1024>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[1],round,leveloff[1],nodeoff[1],S[1],L[1]);
		if(flag2==0)hipMemcpyAsync(m2,dev_m2,sizeof(int),hipMemcpyDeviceToHost,stream1);
		if(flag2==0)hipMemcpyAsync(m1,dev_m1,sizeof(int),hipMemcpyDeviceToHost,stream0);
		if(flag1==0)hipStreamSynchronize(stream1);
		if(*m2==0)flag2=1;
		if(flag1==0)hipStreamSynchronize(stream0);
		if(*m1==0)flag1=1;*/
	}
	//hipStreamSynchronize(stream1);
	//hipStreamSynchronize(stream0);
	hipMemcpy(d,dev_d,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	for(int j=0;j<8;j++)
		{for(int i=0;i<nodenum;i++)
			cout<<d[i+j*nodenum]<<" ";
		cout<<endl;
		}
	cout<<endl;
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	vector<vector<int>>result(LY,vector<int>());
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	hipFree(dev_w);
	cout<<"before return"<<endl;
	return result;
};