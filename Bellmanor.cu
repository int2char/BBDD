#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void Bellmanor::copydata(int s,vector<edge>&edges,int nodenum){
};
void Bellmanor::dellocate(){
};
void Bellmanor::allocate(int maxn,int maxedge){
}
void Bellmanor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void Bellmanor::updatE(vector<int>esigns)
{
	
}
void Bellmanor::updatS(vector<vector<pair<int,int>>>&stpair)
{
	L[0]=LY1;
	L[1]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	int count=0;
	ncount=L[0]*S[0]+L[1]*S[1];
	for(int i=0;i<nodenum*ncount;i++)
		d[i]=INT_MAX/2,p[i]=-1;
	for(int h=0;h<stpair.size();h++)
		for(int k=0;k<L[h];k++)
			{
				int boff=k*S[h]*nodenum;
				for(int i=0;i<S[h];i++)
				{
					int soff=i*nodenum;
					for(int j=0;j<stpair[h].size();j++)
						{
						 d[boff+soff+stpair[h][i].first]=0;
						 has[boff+soff+stpair[h][i].first]=0;
						 count++;
						}
				}
			}
	for(int i=1;i<NF.size();i++)
		NF[i]=L[i-1]*S[i-1];
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void Bellmanor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf):L(2,0),S(2,0),NF(2,0)
{
	/*nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	has=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	w=new int[edges.size()*LY];
	m1=new int;
	m2=new int;
	*m1=0,*m2=0;
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	int cc=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			w[cc++]=esigns[k][i];
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_has,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_w,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_m1,sizeof(int));
	hipMalloc((void**)&dev_m2,sizeof(int));
	if(dev_d==NULL) {
		printf("couldn't allocate %d int's.\n");
	}
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_w,w,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_has,has,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice);
	cout<<nodenum<<endl;*/
};
Bellmanor::Bellmanor()
{
};
__global__ void bellmanhigh(int *st,int *te,int *d,int *has,int *w,int E,int N,int size,int *m,int round,int Leveloff,int numoff,int YE,int LY)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*LY));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*LY))*N+(eid/E)*N*YE+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight<d[t+off])
		{
			d[t+off]=weight+d[s+off];
			has[t+off]=round;
			*m=1;
		}
}
__global__ void color(int *st,int *te,int *d,int *pre,int *has,int *w,int E,int N,int size,int round,int Leveloff,int numoff,int YE,int LY)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*LY));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*LY))*N+(eid/E)*N*YE+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight==d[t+off])
		pre[t+off]=s+off;
}
vector<vector<int>> Bellmanor::routalg(int s,int t,int bw)
{
	int kk=1;
	time_t start,end;
	start=clock();
	int size1=edges.size()*L[0]*S[0];
	cout<<"size is: "<<size<<endl;
	*m=1;
	int round=1;
	cout<<"asdaasadadadasd"<<endl;
	while(*m==1)
	{
		*m=0;
		hipMemcpy(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice);
		bellmanhigh<<<size/1024+1,1024>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size1,dev_m1,round,0,0,S[0],0);
	 	color<<<size/1024+1,1024>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size2,0,0,S[0],0);
		round++;
		hipMemcpy(m1,dev_m2,sizeof(int),hipMemcpyDeviceToHost);
	}
	hipMemcpy(d,dev_d,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	hipStreamSynchronize(0);
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	vector<vector<int>>result(LY,vector<int>());
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	hipFree(dev_w);
	cout<<"before return"<<endl;
	return result;
};