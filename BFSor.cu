#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void BFSor::copydata(int s,vector<edge>&edges,int nodenum){
};
void BFSor::dellocate(){
};
void BFSor::allocate(int maxn,int maxedge){
}
void BFSor::topsort()
{
};
void BFSor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[2*edges.size()*LY];
	te=new int[2*edges.size()*LY];
	d=new int[nodenum*LY*YE];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neibn[t].push_back(s);
			neie[s].push_back(i);
			neie[t].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]==-1)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=WD+1;
	for(int k=0;k<LY;k++)
	{
		int boff=k*YE*nodenum;
		for(int i=0;i<YE;i++)
		{
			int soff=i*nodenum;
			for(int j=0;j<stpair.size();j++)
				d[boff+soff+stpair[i].first]=0;
		}
	}
	hipMalloc((void**)&dev_st,2*LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,2*LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	if(dev_d==NULL) {
		printf("couldn't allocate %d int's.\n");
	}
	hipMemcpy(dev_te,te,2*LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,2*LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	cout<<nodenum<<endl;
};
BFSor::BFSor()
{
	
};
__global__ void BFSfast(int *st,int *te,int *d,int round,int E,int N,int size)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid];
	int off=(i/(E*LY))*N+(eid/E)*N*YE;
	if(d[s+off]==round-1&&d[t+off]>round)
		d[t+off]=round;
}
vector<vector<int>> BFSor::routalg(int s,int t,int bw)
{
	cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=2*edges.size()*LY*YE;
	for(int i=1;i<=WD;i++)
		BFSfast<<<size/512+1,512>>>(dev_st,dev_te,dev_d,i,2*edges.size(),nodenum,size);
	hipMemcpy(d,dev_d,nodenum*LY*YE*sizeof(int),hipMemcpyDeviceToHost);
	hipStreamSynchronize(0);
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	vector<vector<int>>result(LY,vector<int>());
	for(int k=0;k<LY;k++)
	{
		int woff=k*YE*nodenum;
		for(int i=0;i<YE;i++)
		{
			result[k].push_back(d[woff+i*nodenum+stp[i].second]);
		}
	}
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	cout<<"before return"<<endl;
	return result;
};