#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void BFSor::copydata(int s,vector<edge>&edges,int nodenum){
};
void BFSor::dellocate(){
};
void BFSor::allocate(int maxn,int maxedge){
}
void BFSor::topsort()
{
};
void BFSor::updatE(vector<int>esigns)
{
};
void BFSor::updatS(vector<vector<Sot>>&stpair)
{
	L[0]=0;
	L[1]=LY1;
	L[2]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	stps=stpair;
	int count=0;
	ncount=L[1]*S[0]+L[2]*S[1];
	for(int i=0;i<nodenum*ncount;i++)
		d[i]=INF,p[i]=-1;
	int nut=(IFHOP>0)?(WD+1):1;
	for(int h=0;h<stpair.size();h++)
		{
		for(int k=0;k<L[h+1];k++)
			{
			for(int j=0;j<stpair[h].size();j++)
				{
				 d[count*nodenum+stpair[h][j].s*nut]=0;
				 count++;
				}
			}
		}
	Size[0]=edges.size()*L[1]*S[0];
	Size[1]=edges.size()*L[2]*S[1];
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void BFSor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	cout<<"in BFS init"<<endl;
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	stid=new int[edges.size()*LY]
	d=new int[nodenum*LY*YE];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	/*for(int i=0;i<LY;i++)
		{	
			int off=LY*edges.size();
			for(int j=0;j<edges.size();j++)
				esignes[off+i]=esigns[i][j];
		}*/
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				stid[count]=neie[i][j];
				count++;
			}
	
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=WD+1,p[i]=-1;
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_stid,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_esignes,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_stid,stid,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_esignes,esignes,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
};
BFSor::BFSor():L(PC+1,0),S(PC,0),NF(PC,0),Size(2,0)
{
};
__global__ void BFSfast(int *st,int *te,int *d,int *stid,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid];
	if(s==t)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	if(d[s+off]==round-1&&d[t+off]>round)
		{	d[t+off]=round;
			p[t+off]=stid[eeid];
		}
}
vector<vector<Rout>> BFSor::routalg(int s,int t,int bw)
{
	cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=edges.size()*LY*YE;
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	int leoff=edges.size()*L[1];
	int nuoff=L[1]*S[0]*nodenum;
	for(int i=1;i<=WD;i++)
		{
			BFSfast<<<Size[0]/512+1,512,0,stream0>>>(dev_st,dev_te,dev_d,dev_stid,edges.size(),nodenum,Size[0],i,0,0,S[0],L[1]);
			BFSfast<<<Size[1]/512+1,512,0,stream1>>>(dev_st,dev_te,dev_d,dev_stid,edges.size(),nodenum,Size[1],i,leoff,nuoff,S[1],L[2]);
		}
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream0);
	hipMemcpy(d,dev_d,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(p,dev_p,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	vector<vector<Rout>>result(2,vector<Rout>());
	for(int y=1;y<PC+1;y++)
		for(int k=L[y-1];k<L[y];k++)
		{
			int tnode=-1;
			for(int l=0;l<stpairs[y-1].size();l++)
			{	
				int s=stpairs[y-1][l].s*(WD+1);
				vector<int>ters=stpairs[y-1][l].ters;
				for(int i=0;i<ters.size();i++)
				{
					Rout S()
					result 
				}
				
			}
		}
	
	
	
	
	
	
	
	
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	cout<<"before return"<<endl;
	return result;
};
/*__global__ void BFSfast(int *st,int *te,int *d,int round,int E,int N,int size)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid];
	int off=(i/(E*LY))*N+(eid/E)*N*YE;
	if(d[s+off]==round-1&&d[t+off]>round)
		d[t+off]=round;
}*/