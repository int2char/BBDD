#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void BFSor::copydata(int s,vector<edge>&edges,int nodenum){
};
void BFSor::dellocate(){
};
void BFSor::allocate(int maxn,int maxedge){
}
void BFSor::topsort()
{
};
void BFSor::updatE(vector<int>esigns)
{
};
void BFSor::updatS(vector<vector<Sot>>&stpair)
{
	L[0]=0;
	L[1]=LY1;
	L[2]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	stps=stpair;
	int count=0;
	ncount=L[1]*S[0]+L[2]*S[1];
	for(int i=0;i<nodenum*ncount;i++)
		d[i]=INF;
	int nut=(IFHOP>0)?(WD+1):1;
	for(int h=0;h<stpair.size();h++)
		{
		for(int k=0;k<L[h+1];k++)
			{
			for(int j=0;j<stpair[h].size();j++)
				{
				 d[count*nodenum+stpair[h][j].s*nut]=0;
				 count++;
				}
			}
		}
	Size[0]=edges.size()*L[1]*S[0];
	Size[1]=edges.size()*L[2]*S[1];
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void BFSor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	cout<<"in BFS init"<<endl;
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=WD+1;
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_d,d,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
};
BFSor::BFSor():L(PC+1,0),S(PC,0),NF(PC,0),Size(2,0)
{
};
__global__ void BFSfast(int *st,int *te,int *d,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid];
	if(s==t)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	if(d[s+off]==round-1&&d[t+off]>round)
		d[t+off]=round;
}
vector<vector<vector<int>>> BFSor::routalg(int s,int t,int bw)
{
	cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=edges.size()*LY*YE;
	/*for(int i=0;i<8;i++)
		{
			for(int j=0;j<nodenum;j++)
				cout<<d[i*nodenum+j]<<" ";
			cout<<endl;
		}*/
	cout<<"size"<<Size[0]<<endl;
	int leoff=edges.size()*L[1];
	int nuoff=L[1]*S[0]*nodenum;
	for(int i=1;i<=WD;i++)
		{
			BFSfast<<<Size[0]/512+1,512>>>(dev_st,dev_te,dev_d,edges.size(),nodenum,Size[0],i,0,0,S[0],L[1]);
			BFSfast<<<Size[1]/512+1,512>>>(dev_st,dev_te,dev_d,edges.size(),nodenum,Size[1],i,leoff,nuoff,S[1],L[2]);
		}
	hipMemcpy(d,dev_d,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	vector<vector<vector<int>>>result(2,vector<vector<int>>());
	vector<vector<int>>darray1,darray2;
	/*for(int j=0;j<L[0];j++)
	{
		vector<int>res;
		for(int k=0;k<stps[0].size();k++)
			res.push_back(d[j*stps[0].size()*nodenum+k*nodenum]);
		darray1.push_back(res);
	}
	
	for(int j=0;j<L[1];j++)
	{
		vector<int>res;
		for(int k=0;k<stps[1].size();k++)
			res.push_back(d[L[0]*stps[0].size()*nodenum+j*stps[1].size()*nodenum+k*nodenum]);
		darray2.push_back(res);
	}
	result.push_back(darray1);
	result.push_back(darray2);*/
	/*for(int i=0;i<8;i++)
		{
			for(int j=0;j<nodenum;j++)
				cout<<d[i*nodenum+j]<<" ";
			cout<<endl;
		}*/
	hipMemcpy(d,dev_d,nodenum*LY*YE*sizeof(int),hipMemcpyDeviceToHost);
	hipStreamSynchronize(0);
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	cout<<"before return"<<endl;
	return result;
};
/*__global__ void BFSfast(int *st,int *te,int *d,int round,int E,int N,int size)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid];
	int off=(i/(E*LY))*N+(eid/E)*N*YE;
	if(d[s+off]==round-1&&d[t+off]>round)
		d[t+off]=round;
}*/